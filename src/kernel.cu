﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include "Gaas.cuh"
#include "Stopwatch.hpp"
#include "file_io.hpp"


int main()
{
	gaas::lineshapeSim::simHandler sh = gaas::lineshapeSim::simHandler("/home/gputestbed/Desktop/WMS_Processing_V2_ARPAE/gaas/H2Oh2oTest", "/home/gputestbed/Desktop/WMS_Processing_V2_ARPAE/gaas/H2O_tips.csv");
	
	std::cout << "MEMORY TEST: check tops, repeatedly running absorption sim.\n";
	while(1){
		int wavenumRes = (6000-100)*1000;
		float* wavenums = new float[wavenumRes];
		float* spec = new float[wavenumRes];
		sh.runFloat(300, 1, .1, spec, wavenums, wavenumRes, 100, 6000);
		delete [] wavenums;
		delete [] spec;
	}
	
	//fio::save_to_bin_file((char*)spec, wavenumRes * sizeof(float), "/home/gputestbed/Desktop/GAAS/GAAS/Validation/output");
	
	
	return 0;
}


