﻿
#include <stdio.h>
#include <iostream>
#include <map>
#include "hip/hip_runtime.h"
#include ""
#include "Gaas.cuh"
#include "Stopwatch.hpp"
#include "file_io.hpp"



int main()
{
	gaas::lineshapeSim::simHandler sh = gaas::lineshapeSim::simHandler("gaas/H2Oh2oTest", "/gaas/H2O_tips.csv");
	int wavenumRes = (6000-100)*1000;
	float* wavenums = new float[wavenumRes];
	float* spec = new float[wavenumRes];
	sh.runFloat(300, 1, .1, spec, wavenums, wavenumRes, 100, 6000,18.01528,1);
	delete [] wavenums;
	delete [] spec;
	
	
	//fio::save_to_bin_file((char*)spec, wavenumRes * sizeof(float), "/home/gputestbed/Desktop/GAAS/GAAS/Validation/output");
	
	
	return 0;
}


